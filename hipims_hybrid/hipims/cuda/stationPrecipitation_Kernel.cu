#include "hip/hip_runtime.h"
// #include <torch/extension.h>
#include "gpu.cuh"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <torch/extension.h>

template <typename scalar_t>
__global__ void
station_PrecipitationCalculation_kernel(int N, scalar_t *__restrict__ h_update,
                                        int16_t *__restrict__ rainStationMask,
                                        scalar_t *__restrict__ rainStationData,
                                        scalar_t *__restrict__ dt) {
  // get the index of cell
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    h_update[i] += rainStationData[rainStationMask[i]] * dt[0];
  }
}

void station_PrecipitationCalculation_cuda(at::Tensor h_update,
                                           at::Tensor rainStationMask,
                                           at::Tensor rainStationData,
                                           at::Tensor dt) {
  const int N = rainStationMask.numel();
  at::cuda::CUDAGuard device_guard(rainStationMask.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(
      h_update.type(), "station_PrecipitationCalculation", ([&] {
        station_PrecipitationCalculation_kernel<
            scalar_t><<<GET_BLOCKS(N), CUDA_NUM_THREADS, 0, stream>>>(
            N, h_update.data<scalar_t>(), rainStationMask.data<int16_t>(),
            rainStationData.data<scalar_t>(), dt.data<scalar_t>());
      }));
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error in load_textures: %s\n", hipGetErrorString(err));
}
